
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


__global__ void addvecs(float *A_d,float*B_d,float*C_d, int n)
{
    int i=blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n)
    {
        C_d[i]=A_d[i]+B_d[i];
    }
}


int main()
{
    int n=100;
    float *A_h,*B_h,*C_h;
    A_h=new float[n];
    B_h=new float[n];
    C_h=new float[n];
    for(int i=0;i<n;i++)
    {
        A_h[i]=1;
        B_h[i]=2;
        C_h[i]=8;
    }
    float *A_d,*B_d,*C_d;
    hipError_t err;

    err = hipMalloc(&A_d, n * sizeof(float));
    if (err != hipSuccess) { cout << "CUDA malloc failed for A_d: " << hipGetErrorString(err) << endl; return -1; }

    err = hipMalloc(&B_d, n * sizeof(float));
    if (err != hipSuccess) { cout << "CUDA malloc failed for B_d: " << hipGetErrorString(err) << endl; return -1; }

    err = hipMalloc(&C_d, n * sizeof(float));
    if (err != hipSuccess) { cout << "CUDA malloc failed for C_d: " << hipGetErrorString(err) << endl; return -1; }
    hipMemcpy(A_d,A_h,n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,n*sizeof(float),hipMemcpyHostToDevice);

    int threads=256;
    // cout<<ceil(n/256.0)<<endl;
    //return 0;
    addvecs<<<ceil(n/256.0),threads>>>(A_d,B_d,C_d,n);
    hipMemcpy(C_h,C_d,n*sizeof(float),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipError_t errr = hipGetLastError();
    if (errr != hipSuccess) {
        cout << "CUDA Kernel Error: " << hipGetErrorString(errr) << endl;
    }

    cout<<C_h[1]<<endl;
    cout<<A_h[1]<<endl;
    cout<<B_h[1]<<endl;

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    delete [] A_h;
    delete [] B_h;
    delete [] C_h;


}